#include "hip/hip_runtime.h"
#include <iostream>
#include "hello.h"

__global__ void sayHello() {
    printf("Hello world from the GPU!\n");
}

int cudaHello(mythread::Problem& problem) {
    printf("Hello world from the CPU!\n");
    uint index;
    double x, y;
    problem.getProblemData(index, x, y);
    std::cout << "This problem is -> index: " << index << " (x,y): " << "(" << x << "," << y << ")\n";

    sayHello<<<1,1>>>();
    hipDeviceSynchronize();

    return 0;
}

int makeMatrix(){
    int   num_rows         = 5;
    int   num_cols         = 4;
    int   nnz              = 11;
    int   ld               = num_cols;
    int   dense_size       = ld * num_rows;
    int   h_csr_offsets[]  = { 0, 3, 4, 7, 9, 11 };
    int   h_csr_columns[]  = { 0, 2, 3, 1, 0, 2, 3, 1, 3, 1, 2 };
    float h_csr_values[]   = { 1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f,
                               7.0f, 8.0f, 9.0f, 10.0f, 11.0f };
    float h_dense[]        = { 0.0f, 0.0f, 0.0f, 0.0f,
                               0.0f, 0.0f, 0.0f, 0.0f,
                               0.0f, 0.0f, 0.0f, 0.0f,
                               0.0f, 0.0f, 0.0f, 0.0f,
                               0.0f, 0.0f, 0.0f, 0.0f };
    float h_dense_result[] = { 1.0f,  0.0f,  2.0f,  3.0f,
                               0.0f,  4.0f,  0.0f,  0.0f,
                               5.0f,  0.0f,  6.0f,  7.0f,
                               0.0f,  8.0f,  0.0f,  9.0f,
                               0.0f, 10.0f, 11.0f,  0.0f };

    //--------------------------------------------------------------------------
    // Device memory management
    int   *d_csr_offsets, *d_csr_columns;
    float *d_csr_values,  *d_dense;
    CHECK_CUDA( hipMalloc((void**) &d_csr_offsets,
                           (num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &d_csr_columns, nnz * sizeof(int))         )
    CHECK_CUDA( hipMalloc((void**) &d_csr_values,  nnz * sizeof(float))       )
    CHECK_CUDA( hipMalloc((void**) &d_dense,       dense_size * sizeof(float)))

    CHECK_CUDA( hipMemcpy(d_csr_offsets, h_csr_offsets,
                           (num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(d_csr_columns, h_csr_columns, nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(d_csr_values, h_csr_values, nnz * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(d_dense, h_dense, dense_size * sizeof(float),
                           hipMemcpyHostToDevice) )
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )

    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(d_csr_offsets) )
    CHECK_CUDA( hipFree(d_csr_columns) )
    CHECK_CUDA( hipFree(d_csr_values) )
    CHECK_CUDA( hipFree(d_dense) )
    return EXIT_SUCCESS;
}